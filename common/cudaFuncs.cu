#include <cstdio>
#include "hipfft/hipfft.h"

#include "cudaFuncs.cuh"
#include "logger.hpp"

int cuda_count_devices() {
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);

    return nDevices;
}

void cuda_list_devices() {
    int nDevices = cuda_count_devices();

    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        DispInfo("Birch", "Compute device %d: %s", i + 1, prop.name);
    }
}

void cuda_init() {
    hipfftHandle handle;
    hipfftPlan1d(&handle, 2048, hipfftType_t::HIPFFT_Z2Z, 1);

    hipfftDestroy(handle);
}

void cuda_get_device_memory(int gpu, size_t *free, size_t *total) {
    hipSetDevice(gpu);
    hipMemGetInfo(free, total);
}