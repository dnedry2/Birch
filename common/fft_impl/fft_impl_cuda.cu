#include "fft.h"

#include <cstdio>
#include "hipfft/hipfft.h"

using namespace Birch;

struct cudaPlan {
    hipfftHandle handle;

    hipfftDoubleComplex* inBuf  = nullptr;
    hipfftDoubleComplex* outBuf = nullptr;

    const unsigned size = 0;
    const unsigned cnt  = 0;
    const unsigned bufferBytes = 0;
    const unsigned device = 0;

    cudaPlan(unsigned size, unsigned cnt, unsigned dev) : size(size), cnt(cnt), bufferBytes(size * cnt * sizeof(*outBuf)), device(dev) { }
};

fft_gpu_plan fft_cuda_make_plan(unsigned maxSize, unsigned fftCnt, unsigned dev) {
    hipSetDevice(dev);

    cudaPlan* plan = new cudaPlan(maxSize, fftCnt, dev);

    hipfftPlan1d(&plan->handle, maxSize, hipfftType_t::HIPFFT_Z2Z, fftCnt);
    hipMalloc((void**)&plan->inBuf,  plan->bufferBytes);
    hipMalloc((void**)&plan->outBuf, plan->bufferBytes);

    return static_cast<fft_gpu_plan>(plan);
}
void fft_cuda_destroy_plan(fft_gpu_plan plan) {
    cudaPlan* const cPlan = static_cast<cudaPlan*>(plan);

    hipSetDevice(cPlan->device);

    hipfftDestroy(cPlan->handle);
    hipFree(cPlan->inBuf);
    hipFree(cPlan->outBuf);

    delete cPlan;
}

void fft_cuda_cpx_forward(fft_gpu_plan plan, Complex<double>* input, Complex<double>* output) {
    cudaPlan* const cPlan = static_cast<cudaPlan*>(plan);
    hipSetDevice(cPlan->device);

    hipMemcpy((void*)cPlan->inBuf, (void*)input, cPlan->bufferBytes, hipMemcpyHostToDevice);

    hipfftExecZ2Z(cPlan->handle, cPlan->inBuf, cPlan->outBuf, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    hipMemcpy((void*)output, (void*)cPlan->outBuf, cPlan->bufferBytes, hipMemcpyDeviceToHost);
}
void fft_cuda_cpx_inverse(fft_gpu_plan plan, Complex<double>* input, Complex<double>* output) {
    cudaPlan* const cPlan = static_cast<cudaPlan*>(plan);
    hipSetDevice(cPlan->device);

    hipMemcpy((void*)cPlan->inBuf, (void*)input, cPlan->bufferBytes, hipMemcpyHostToDevice);

    hipfftExecZ2Z(cPlan->handle, cPlan->inBuf, cPlan->outBuf, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    hipMemcpy((void*)output, (void*)cPlan->outBuf, cPlan->bufferBytes, hipMemcpyDeviceToHost);
}