#include <hip/hip_runtime.h>

__global__ void calcDTOAKernel(double* out, const double* in, const bool* mask, unsigned long elCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < elCount)
        if (mask == nullptr || mask[i])
            out[i] = in[i] - in[i - 1];
}

void calcDTOACUDA(double* out, const double* in, const bool* mask, unsigned long elCount) {
    hipSetDevice(0);

    int threadsPerBlock = 256;
    int blocksPerGrid = (elCount + threadsPerBlock - 1) / threadsPerBlock;

    // Copy data to device
    double* d_in;
    double* d_out;
    bool*   d_mask;

    hipMalloc(&d_in,   elCount * sizeof(double));
    hipMalloc(&d_out,  elCount * sizeof(double));

    if (mask != nullptr)
        hipMalloc(&d_mask, elCount * sizeof(bool));

    hipMemcpy(d_in,   in,   elCount * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, elCount * sizeof(bool),   hipMemcpyHostToDevice);
    
    calcDTOAKernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_in, mask == nullptr ? nullptr : d_mask, elCount);
    hipDeviceSynchronize();

    // Copy data back to host
    hipMemcpy(out, d_out, elCount * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_in);
    hipFree(d_out);

    if (mask != nullptr)
        hipFree(d_mask);
}